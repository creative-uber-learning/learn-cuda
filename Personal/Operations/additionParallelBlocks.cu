
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 512


void random_ints(int* a, int n)
{
   int i;
   for (i = 0; i < n; ++i) {
       a[i] = rand() %5000;
   }
}

// each parallel invocation of add() is referred to as a block : the set of blocks is referred to as a grid
__global__ void add(int *a, int *b, int *c)
{
    // each block handles a different element of the array
    // on the device, each block can execute in parallel
    // use blockIdx.x to access block index
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main(void)
{
    // host copies of a, b, c
    int *a, *b, *c; 
    // device copies of a, b, c
    int *d_a, *d_b, *d_c; 
    int size = N * sizeof(int);

    // we need to allocate memory on the GPU
    // allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // allocate space for host copies of a, b, c and setup input values
    a = (int *)malloc(size); 
    random_ints(a, N);
    b = (int *)malloc(size); 
    random_ints(b, N);
    c = (int *)malloc(size);

    // copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // launch add() kernel on the GPU with N blocks
    add<<<N,1>>>(d_a, d_b, d_c);

    // copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // don't forget to free the memory
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // check error 
    printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));

    return 0;
}